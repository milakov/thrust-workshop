
#include <hip/hip_runtime.h>
#include <vector>
#include <cstdlib>
#include <iostream>


// TODO: annotate this function with __host__ __device__ so
//       so that it is able to work with Thrust
float2 operator+(float2 a, float2 b)
{
  return make_float2(a.x + b.x, a.y + b.y);
}


// given an integer, output a pseudorandom 2D point
struct random_point
{
  __host__ __device__ unsigned int hash(unsigned int x)
  {
    x = (x+0x7ed55d16) + (x<<12);
    x = (x^0xc761c23c) ^ (x>>19);
    x = (x+0x165667b1) + (x<<5);
    x = (x+0xd3a2646c) ^ (x<<9);
    x = (x+0xfd7046c5) + (x<<3);
    x = (x^0xb55a4f09) ^ (x>>16);
    return x;
  }

  __host__ __device__
  float2 operator()(unsigned int x)
  {
    return make_float2(float(hash(x)) / UINT_MAX, float(hash(2 * x)) / UINT_MAX);
  }
};


void generate_random_points(std::vector<float2> &points)
{
  // sequentially generate some random 2D points in the unit square
  std::cout << "TODO: parallelize this loop using thrust::tabulate\n" << std::endl;

  for(int i = 0; i < points.size(); ++i)
  {
    float x = float(rand()) / RAND_MAX;
    float y = float(rand()) / RAND_MAX;

    points[i] = make_float2(x,y);
  }
}


float2 compute_centroid(const std::vector<float2> &points)
{
  float2 sum = make_float2(0,0);

  // compute the sum
  std::cout << "TODO: parallelize this sum using thrust::reduce\n" << std::endl;
  for(int i = 0; i < points.size(); ++i)
  {
    sum = sum + points[i];
  }

  // divide the sum by the number of points
  return make_float2(sum.x / points.size(), sum.y / points.size());
}


// given a 2D point, return which quadrant it is in
struct classify_point
{
  float2 center;

  __host__ __device__
  classify_point(float2 c)
  {
    center = c;
  }

  __host__ __device__
  unsigned int operator()(float2 p)
  {
    return (p.x <= center.x ? 0 : 1) | (p.y <= center.y ? 0 : 2);
  }
};


void classify_points_by_quadrant(const std::vector<float2> &points, float2 centroid, std::vector<int> &quadrants)
{
  // classify each point relative to the centroid
  std::cout << "TODO: parallelize this loop using thrust::transform\n" << std::endl;
  for(int i = 0; i < points.size(); ++i)
  {
    float x = points[i].x;
    float y = points[i].y;

    // bottom-left:  0
    // bottom-right: 1
    // top-left:     2
    // top-right:    3

    quadrants[i] = (x <= centroid.x ? 0 : 1) | (y <= centroid.y ? 0 : 2);
  }
}


void count_points_in_quadrants(std::vector<int> &quadrants, std::vector<int> &counts_per_quadrant)
{
  // sequentially compute a histogram
  std::cout << "TODO: parallelize this loop by" << std::endl;
  std::cout << "   1. sorting quadrant numbers" << std::endl;
  std::cout << "   2. reducing quadrant numbers by quadrant\n" << std::endl;
  for(int i = 0; i < quadrants.size(); ++i)
  {
    int q = quadrants[i];

    // increment the number of points in this quadrant
    counts_per_quadrant[q]++;
  }
}


std::ostream &operator<<(std::ostream &os, float2 p)
{
  return os << "(" << p.x << ", " << p.y << ")";
}


int main()
{
  const size_t num_points = 10000000;

  std::cout << "TODO: move these points to the GPU using thrust::device_vector\n" << std::endl;
  std::vector<float2> points(num_points);

  generate_random_points(points);

  float2 centroid = compute_centroid(points);

  std::cout << "TODO: move these quadrants to the GPU using thrust::device_vector\n" << std::endl;
  std::vector<int> quadrants(points.size());
  classify_points_by_quadrant(points, centroid, quadrants);

  std::cout << "TODO: move these counts to the GPU using thrust::device_vector\n" << std::endl;
  std::vector<int> counts_per_quadrant(4);
  count_points_in_quadrants(quadrants, counts_per_quadrant);

  std::cout << "Per-quadrant counts:" << std::endl;
  std::cout << "  Bottom-left : " << counts_per_quadrant[0] << " points" << std::endl;
  std::cout << "  Bottom-right: " << counts_per_quadrant[1] << " points" << std::endl;
  std::cout << "  Top-left    : " << counts_per_quadrant[2] << " points" << std::endl;
  std::cout << "  Top-right   : " << counts_per_quadrant[3] << " points" << std::endl;
  std::cout << std::endl;
}

